#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
/**
 * CUDA MD5 cracker
 * Copyright (C) 2015  Konrad Kusnierz <iryont@gmail.com>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <stdio.h>
#include <iostream>
#include <time.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>

#define CONST_WORD_LIMIT 10
#define CONST_CHARSET_LIMIT 100

#define CONST_CHARSET "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789"
#define CONST_CHARSET_LENGTH (sizeof(CONST_CHARSET) - 1)

#define CONST_WORD_LENGTH_MIN 1
#define CONST_WORD_LENGTH_MAX 8
#define HASHES_PER_KERNEL 1 //128UL

#include "assert.cu"
#include "md5.cu"
#include "sha1.cu"

 /* Global variables */
uint8_t g_wordLength;

char g_word[CONST_WORD_LIMIT];
char g_charset[CONST_CHARSET_LIMIT];
char g_cracked[CONST_WORD_LIMIT];
int BLOCKS, THREADS;

__device__ char g_deviceCharset[CONST_CHARSET_LIMIT];
__device__ char g_deviceCracked[CONST_WORD_LIMIT];

__device__ __host__ bool next(uint8_t* length, char* word, uint32_t increment) {
    uint32_t idx = 0;
    uint32_t add = 0;

    while (increment > 0 && idx < CONST_WORD_LIMIT) {
        if (idx >= *length && increment > 0) {
            increment--;
        }

        add = increment + word[idx];
        word[idx] = add % CONST_CHARSET_LENGTH;
        increment = add / CONST_CHARSET_LENGTH;
        idx++;
    }

    if (idx > * length) {
        *length = idx;
    }

    if (idx > CONST_WORD_LENGTH_MAX) {
        return false;
    }

    return true;
}

__global__ void md5Crack(uint8_t wordLength, char* charsetWord, uint32_t hash01, uint32_t hash02, uint32_t hash03, uint32_t hash04) {
    uint32_t idx = (blockIdx.x * blockDim.x + threadIdx.x) * HASHES_PER_KERNEL;

    /* Shared variables */
    __shared__ char sharedCharset[CONST_CHARSET_LIMIT];

    /* Thread variables */
    char threadCharsetWord[CONST_WORD_LIMIT];
    char threadTextWord[CONST_WORD_LIMIT];
    uint8_t threadWordLength;
    uint32_t threadHash01, threadHash02, threadHash03, threadHash04;

    /* Copy everything to local memory */
    memcpy(threadCharsetWord, charsetWord, CONST_WORD_LIMIT);
    memcpy(&threadWordLength, &wordLength, sizeof(uint8_t));
    memcpy(sharedCharset, g_deviceCharset, sizeof(uint8_t) * CONST_CHARSET_LIMIT);

    /* Increment current word by thread index */
    next(&threadWordLength, threadCharsetWord, idx);

    for (uint32_t hash = 0; hash < HASHES_PER_KERNEL; hash++) {
        for (uint32_t i = 0; i < threadWordLength; i++) {
            threadTextWord[i] = sharedCharset[threadCharsetWord[i]];
        }

        md5Hash((unsigned char*)threadTextWord, threadWordLength, &threadHash01, &threadHash02, &threadHash03, &threadHash04);

        if (threadHash01 == hash01 && threadHash02 == hash02 && threadHash03 == hash03 && threadHash04 == hash04) {
            memcpy(g_deviceCracked, threadTextWord, threadWordLength);
        }

        if (!next(&threadWordLength, threadCharsetWord, 1)) {
            break;
        }
    }
}


__global__ void sha1Crack(uint8_t wordLength, char* charsetWord, uint32_t hash01, uint32_t hash02, uint32_t hash03, uint32_t hash04, uint32_t hash05) {
    uint32_t idx = (blockIdx.x * blockDim.x + threadIdx.x) * HASHES_PER_KERNEL;

    /* Shared variables */
    __shared__ char sharedCharset[CONST_CHARSET_LIMIT];

    /* Thread variables */
    char threadCharsetWord[CONST_WORD_LIMIT];
    char threadTextWord[CONST_WORD_LIMIT];
    uint8_t threadWordLength;
    uint32_t threadHash01, threadHash02, threadHash03, threadHash04, threadHash05;

    /* Copy everything to local memory */
    memcpy(threadCharsetWord, charsetWord, CONST_WORD_LIMIT);
    memcpy(&threadWordLength, &wordLength, sizeof(uint8_t));
    memcpy(sharedCharset, g_deviceCharset, sizeof(uint8_t) * CONST_CHARSET_LIMIT);

    /* Increment current word by thread index */
    next(&threadWordLength, threadCharsetWord, idx);
    //printf("%d", wordLength);
    for (uint32_t hash = 0; hash < HASHES_PER_KERNEL; hash++) {
        for (uint32_t i = 0; i < wordLength; i++) {
            threadTextWord[i] = sharedCharset[threadCharsetWord[i]];
        }

        sha1((unsigned char*)threadTextWord, wordLength, &threadHash01, &threadHash02, &threadHash03, &threadHash04, &threadHash05);
        //printf("%s (%d) :: %x\t%x\t%x\t%x\t%x\n", threadTextWord, wordLength, threadHash01, threadHash02, threadHash03, threadHash04, threadHash05);
        if (threadHash01 == hash01 && threadHash02 == hash02 && threadHash03 == hash03 && threadHash04 == hash04 && threadHash05 == hash05) {
            memcpy(g_deviceCracked, threadTextWord, wordLength);
        }

        if (!next(&threadWordLength, threadCharsetWord, 1)) {
            break;
        }
    }
    //printf("AFTER %d", wordLength);
}


int main(int argc, char* argv[]) {
    /* Check arguments */
    //if (argc != 2 || strlen(argv[1]) != 32) {
    //    std::cout << argv[0] << " <md5_hash>" << std::endl;
    //    return -1;
    //}

    /* Amount of available devices */
    int devices;
    ERROR_CHECK(hipGetDeviceCount(&devices));
    hipDeviceProp_t deviceProp;
    int* prop[2] = { 0 , 0 };


    for (int i = 0; i < devices; i++)
    {
        if (hipSuccess != hipGetDeviceProperties(&deviceProp, i))
        {
            BLOCKS += 64;
            THREADS += 128;
            return 0;
        }
        BLOCKS += deviceProp.multiProcessorCount;
        THREADS += deviceProp.maxThreadsPerBlock;
    }

    /* Sync type */
    ERROR_CHECK(hipSetDeviceFlags(hipDeviceScheduleSpin));

    /* Display amount of devices */
    std::cout << "|**********************/" << std::endl;
    std::cout << "|    " << devices << " device(s) found" << std::endl;
    std::cout << "|    " << BLOCKS << " blocks found" << std::endl;
    std::cout << "|    " << THREADS << " threads found" << std::endl;
    std::cout << "|**********************/" << std::endl;



    /* Hash stored as u32 integers */
    //uint32_t md5Hash[4];
    //md5
    //char* hash = "1c0d894f6f6ab511099a568f6e876c2f";

    //sha1
    char* hash = "3e9d6d9f0fd38a6f3e59c5df2f274afed24d0b2f";

    /* Parse argument (md5)*/
    //for (uint8_t i = 0; i < 4; i++) {
    //    char tmp[16];
    //    strncpy(tmp, hash + i * 8, 8);
    //    sscanf(tmp, "%x", &md5Hash[i]);
    //    md5Hash[i] = (md5Hash[i] & 0xFF000000) >> 24 | (md5Hash[i] & 0x00FF0000) >> 8 | (md5Hash[i] & 0x0000FF00) << 8 | (md5Hash[i] & 0x000000FF) << 24;
    //}

    /* Parse argument (sha1)*/
    uint32_t sha1Hash[5];

    char tmp[40];
    for (int i = 0; i < 5; i++)
    {
        for (int j = 0; j < 8; j++)
            tmp[j] = hash[i * 8 + j];

        sha1Hash[i] = (uint32_t)strtoll(tmp, NULL, 16);
    }

    /* Fill memory */
    memset(g_word, 0, CONST_WORD_LIMIT);
    memset(g_cracked, 0, CONST_WORD_LIMIT);
    memcpy(g_charset, CONST_CHARSET, CONST_CHARSET_LENGTH);

    /* Current word length = minimum word length */
    g_wordLength = CONST_WORD_LENGTH_MIN;

    /* Main device */
    hipSetDevice(0);

    /* Time */
    hipEvent_t clockBegin;
    hipEvent_t clockLast;

    hipEventCreate(&clockBegin);
    hipEventCreate(&clockLast);
    hipEventRecord(clockBegin, 0);

    /* Current word is different on each device */
    char** words = new char* [devices];

    for (int device = 0; device < devices; device++) {
        hipSetDevice(device);

        /* Copy to each device */
        ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCharset), g_charset, sizeof(uint8_t) * CONST_CHARSET_LIMIT, 0, hipMemcpyHostToDevice));
        ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCracked), g_cracked, sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyHostToDevice));

        /* Allocate on each device */
        ERROR_CHECK(hipMalloc((void**)&words[device], sizeof(uint8_t) * CONST_WORD_LIMIT));
    }

    while (true) {
        bool result = false;
        bool found = false;

        for (int device = 0; device < devices; device++) {
            hipSetDevice(device);

            /* Copy current data */
            ERROR_CHECK(hipMemcpy(words[device], g_word, sizeof(uint8_t) * CONST_WORD_LIMIT, hipMemcpyHostToDevice));

            /* Start kernel */
            sha1Crack << < BLOCKS, THREADS >> > (g_wordLength, words[device], sha1Hash[0], sha1Hash[1], sha1Hash[2], sha1Hash[3], sha1Hash[4]);

            /* Global increment */
            result = next(&g_wordLength, g_word, BLOCKS * HASHES_PER_KERNEL * THREADS);
        }

        /* Display progress */
        char word[CONST_WORD_LIMIT];

        for (int i = 0; i < g_wordLength; i++) {
            word[i] = g_charset[g_word[i]];
        }

        std::cout << "currently at " << std::string(word, g_wordLength) << " (" << (uint32_t)g_wordLength << ")" << std::endl;

        for (int device = 0; device < devices; device++) {
            hipSetDevice(device);

            /* Synchronize now */
            hipDeviceSynchronize();

            /* Copy result */
            ERROR_CHECK(hipMemcpyFromSymbol(g_cracked, HIP_SYMBOL(g_deviceCracked), sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyDeviceToHost));

            /* Check result */
            if (found = *g_cracked != 0) {
                std::cout << "cracked " << g_cracked << std::endl;
                break;
            }
        }

        if (!result || found) {
            if (!result && !found) {
                std::cout << "found nothing (host)" << std::endl;
            }

            break;
        }
    }

    for (int device = 0; device < devices; device++) {
        hipSetDevice(device);

        /* Free on each device */
        hipFree((void**)words[device]);
    }

    /* Free array */
    delete[] words;

    /* Main device */
    hipSetDevice(0);

    float milliseconds = 0;

    hipEventRecord(clockLast, 0);
    hipEventSynchronize(clockLast);
    hipEventElapsedTime(&milliseconds, clockBegin, clockLast);

    std::cout << "Computation time " << milliseconds << " ms" << std::endl;

    hipEventDestroy(clockBegin);
    hipEventDestroy(clockLast);
}
